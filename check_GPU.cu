#include <iostream>
#include "hip/hip_runtime.h"


int main()
{
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量: " << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个SM的最大线程数: " << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数: " << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}

/* 输出结果为:
使用GPU device 0: NVIDIA A100-PCIE-40GB
SM的数量: 108
每个线程块的共享内存大小：48 KB
每个线程块的最大线程数：1024
每个SM的最大线程数: 2048
每个SM的最大线程束数: 64
*/