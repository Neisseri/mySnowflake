#include "function.h"
#include "tools.h"
int main(int argc,char* argv[]){
    highprecision *phi,*phi_lap,*tempr,*tempr_lap,*phidx,*phidy,*epsilon,*epsilon_deri;
    CHECK_ERROR(hipMallocManaged((void**)&phi,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phi_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&tempr_lap,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidx,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&phidy,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon,sizeof(highprecision)*dimX*dimY));
    CHECK_ERROR(hipMallocManaged((void**)&epsilon_deri,sizeof(highprecision)*dimX*dimY));
    // 分配大小为 dimX * dimY 的二维数组空间
    dim3 blocks(unitx,unity);
    // 二维线程网络中的线程块
    dim3 grids(1,1,unitdimX*unitdimY);
    // 线程网络维度
    dataprepare_high<<<grids,blocks>>>(phi); // 启动 CUDA kernal
    // dataprepare_high 函数定义在 tools.h 中
    hipDeviceSynchronize(); // CPU 等待 GPU 上所有操作全都完成
    
    #ifdef End2end
        hipEvent_t start,stop;float elapsed;
        // start, stop : 记录CUDA事件, 用于测量GPU代码执行时间
        // elapsed : 存储代码执行时间
    #endif
    for(int i=0;i<timesteps;i++){
        #ifdef End2end
            if(i==5){
                CHECK_ERROR(hipEventCreate(&start));
                CHECK_ERROR(hipEventCreate(&stop)); // 创建 CUDA 事件
                CHECK_ERROR(hipEventRecord(start,0));
                // 记录 start 事件时间戳到 stream 0 中
            }
        #endif
        kernel1_pure<<<grids,blocks>>>(phi,phi_lap,tempr,tempr_lap,phidx,phidy,epsilon,epsilon_deri);
        kernel2_pure<<<grids,blocks>>>(phi,phi_lap,epsilon,epsilon_deri,phidx,phidy,tempr,tempr_lap);
        hipDeviceSynchronize();
    }
    #ifdef End2end
        if(timesteps>5){
            CHECK_ERROR(hipEventRecord(stop,0));
            CHECK_ERROR(hipEventSynchronize(stop));
            // 确保之前记录的 CUDA 事件同步完成
            CHECK_ERROR(hipEventElapsedTime(&elapsed,start,stop));
            // 计算时间间隔
            CHECK_ERROR(hipEventDestroy(start));
            CHECK_ERROR(hipEventDestroy(stop));
            // 销毁 CUDA 事件
        }
        ofstream ftime("time_tmp.csv"); // 创建输出文件流对象
        ftime<<elapsed; // 数据计算时间(ms)
        ftime.close();
    #endif
    #ifdef End2end
        if(string(argv[1])=="4"){
            string path=string(argv[2]);
            writetocsv(path,phi,dimX,dimY);
        }
    #endif
    
    CHECK_ERROR(hipFree(phi));
    CHECK_ERROR(hipFree(phi_lap));
    CHECK_ERROR(hipFree(tempr));
    CHECK_ERROR(hipFree(tempr_lap));
    CHECK_ERROR(hipFree(phidx));
    CHECK_ERROR(hipFree(phidy));
    CHECK_ERROR(hipFree(epsilon));
    CHECK_ERROR(hipFree(epsilon_deri));
    // 释放 GPU 内存
    return 0;
}